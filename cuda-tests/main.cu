
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>

int main(void) {
    // measure creation of vector
    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
    const size_t nElements = 8192;
    int* array = new int[nElements];
    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    std::cout << "Time difference = " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << "[µs]" << std::endl; 

    // copy vector to gpu
    begin = std::chrono::steady_clock::now();
    int* gpuArray = NULL;
    auto err = hipMalloc(&gpuArray, sizeof(int) * nElements);
    end = std::chrono::steady_clock::now();
    std::cout << "Time difference = " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << "[µs]" << std::endl; 

    begin = std::chrono::steady_clock::now();
    hipMemcpy(gpuArray, array, sizeof(int) * nElements, ::hipMemcpyHostToDevice);
    end = std::chrono::steady_clock::now();
    std::cout << "Time difference = " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << "[µs]" << std::endl;

}